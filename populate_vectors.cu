#include <math.h>
#include <hip/hip_runtime.h>

__global__ void apply_f1(double h, int lower_bound, double* destination) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	destination[thread_id] = sin(h * (thread_id + lower_bound));
}

__global__ void apply_f2(double h, int lower_bound, double* destination) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	destination[thread_id] = cos(h * (thread_id + lower_bound));
}

extern "C" void populate_vectors(double* f1, double* f2, unsigned int local_length, double h, unsigned int lower_bound) {
	double* f1_d;
	double* f2_d;
	size_t bytes = sizeof(double) * local_length;
	size_t threads = 1024;
	size_t blocks = local_length / threads;
	hipMalloc(&f1_d, bytes);
	hipMalloc(&f2_d, bytes);
	apply_f1<<<blocks,threads>>>(h,lower_bound,f1_d);
	apply_f2<<<blocks,threads>>>(h,lower_bound,f2_d);
	hipDeviceSynchronize();
	hipMemcpy(f1, f1_d, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(f2, f2_d, bytes, hipMemcpyDeviceToHost);
}
